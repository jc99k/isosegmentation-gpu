#include "hip/hip_runtime.h"
#include "Delaunay2D.h"

CDelaunay_2D_Cuda_XGraph_Adaptor::~CDelaunay_2D_Cuda_XGraph_Adaptor()
{
	hipFree(m_pointcoords);
	hipFree(m_cellvertices);
	//hipFree(m_nodeedges);
	//hipFree(m_updatepatterns);
	hipFree(m_sortedcells);
	hipFree(m_colorpatterns);
	hipFree(m_labels);
	hipFree(m_sizes);
	//hipFree(m_facetvertices);
	//hipFree(m_edgenodes);
	//hipFree(m_similarities);

	hipFree(m_neighbors);
	hipFree(m_similarities);
	FreeTexture();
}

void CDelaunay_2D_Cuda_XGraph_Adaptor::UnifiedMalloc()
{
	hipMallocManaged(&m_pointcoords, (dv)*sizeof(PointCoordsType));
	hipMallocManaged(&m_cellvertices, (v)*sizeof(CellVerticesType));
	//hipMallocManaged(&m_nodeedges, (v)*sizeof(NodeEdgesType));
	//hipMallocManaged(&m_updatepatterns, (v)*sizeof(int));
	hipMallocManaged(&m_sortedcells, (v)*sizeof(int));
	hipMallocManaged(&m_colorpatterns, (v)*sizeof(float));
	hipMallocManaged(&m_labels, (v)*sizeof(int));
	hipMallocManaged(&m_sizes, (v)*sizeof(float));
	//hipMallocManaged(&m_facetvertices, (e)*sizeof(FacetsType));
	//hipMallocManaged(&m_edgenodes, (e)*sizeof(EdgeNodesType));
	//hipMallocManaged(&m_similarities, (e)*sizeof(float));
	
	hipMallocManaged(&m_neighbors, (v)*sizeof(NeighborsType));
	hipMallocManaged(&m_similarities, (v)*sizeof(SimilarityType));
}

void CDelaunay_2D_Cuda_XGraph_Adaptor::Synchronize()
{
	hipDeviceSynchronize();
}

void CDelaunay_2D_Cuda_XGraph_Adaptor::SetTexture(ImgType* imgdata, int width, int height)
{
	m_image_dimensions = make_int3(width, height, 1);
	m_vol = width*height;
	size_t pitch, widthInBytes = width*sizeof(ImgType);

	// Allocate pitched (padded) memory in device for texture memory
	// Then copy image data from host into pitched memory, respecting the padding
	hipMallocPitch(&m_texdata, &pitch, widthInBytes, height);
	hipMemcpy2D(m_texdata, pitch, imgdata, widthInBytes, widthInBytes, height, hipMemcpyHostToDevice);

	// Texture (new way) : Create texture object
	// 1. Resource Descriptor (resource to texture with)
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = m_texdata;
	resDesc.res.pitch2D.desc = hipCreateChannelDesc<ImgType>();
	resDesc.res.pitch2D.width = width;
	resDesc.res.pitch2D.height = height;
	resDesc.res.pitch2D.pitchInBytes = pitch;

	// 2. Texture Descriptor (texturing settings)
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
	texDesc.filterMode = hipFilterModeLinear;      // linear interpolation
	texDesc.addressMode[0] = hipAddressModeBorder;
	texDesc.addressMode[1] = hipAddressModeBorder;
	texDesc.readMode = hipReadModeElementType;

	// 3. Create Texture Object (with descriptors)
	hipCreateTextureObject(&m_texObj, &resDesc, &texDesc, NULL);

	//printf("Texture memory ready!\n");
}

void CDelaunay_2D_Cuda_XGraph_Adaptor::FreeTexture()
{
	hipDestroyTextureObject(m_texObj);
	hipFree(m_texdata);
}

const int CDelaunay_2D_Cuda_XGraph_Adaptor::Dim = 2;
const int CDelaunay_2D_Cuda_XGraph_Adaptor::NCellVertices = 3;
const int CDelaunay_2D_Cuda_XGraph_Adaptor::NCellNeighbors = 3;
const int CDelaunay_2D_Cuda_XGraph_Adaptor::NFacetVertices = 2;
const int CDelaunay_2D_Cuda_XGraph_Adaptor::NGraphColors = 4;