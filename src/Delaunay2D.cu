#include "hip/hip_runtime.h"
#include "Delaunay2D.cuh"

CDelaunay_2D_Cuda_XGraph_Adaptor::~CDelaunay_2D_Cuda_XGraph_Adaptor()
{
	hipFree(m_xadj);
	hipFree(m_adjncy);
	hipFree(m_updatepatterns);
	hipFree(m_labels);
	hipFree(m_colorpatterns);
	hipFree(m_similarities);
	hipFree(m_cellvertices);
	hipFree(m_delaunayvertices);

	hipDestroyTextureObject(m_texObj);
	hipFree(m_imagedata);
}

void CDelaunay_2D_Cuda_XGraph_Adaptor::UnifiedMalloc()
{
	hipMallocManaged(&m_xadj, (v + 1)*sizeof(int));
	hipMallocManaged(&m_adjncy, (e * 2)*sizeof(int));
	hipMallocManaged(&m_updatepatterns, (v)*sizeof(int));
	hipMallocManaged(&m_labels, (v)*sizeof(int));
	hipMallocManaged(&m_colorpatterns, (v)*sizeof(float));
	hipMallocManaged(&m_similarities, (e * 2)*sizeof(float));
	hipMallocManaged(&m_cellvertices, (v)*sizeof(int3));
	hipMallocManaged(&m_delaunayvertices, (dv)*sizeof(float2));
}

void CDelaunay_2D_Cuda_XGraph_Adaptor::Synchronize()
{
	hipDeviceSynchronize();
}

void CDelaunay_2D_Cuda_XGraph_Adaptor::SetTexture(ImgType* imgdata, int width, int height)
{
	m_image_dimensions = make_int3(width, height, 1);
	size_t pitch, widthInBytes = width*sizeof(ImgType);

	hipMallocPitch(&m_imagedata, &pitch, widthInBytes, height);
	hipMemcpy2D(m_imagedata, pitch, imgdata, widthInBytes, widthInBytes, height, hipMemcpyHostToDevice);

	// Texture (new way) : Create texture object
	// 1. Resource Descriptor (resource to texture with)
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = m_imagedata;
	resDesc.res.pitch2D.desc = hipCreateChannelDesc<ImgType>();
	resDesc.res.pitch2D.width = width;
	resDesc.res.pitch2D.height = height;
	resDesc.res.pitch2D.pitchInBytes = pitch;

	// 2. Texture Descriptor (texturing settings)
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
	texDesc.filterMode = hipFilterModeLinear;      // linear interpolation
	texDesc.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
	texDesc.addressMode[1] = hipAddressModeWrap;

	// 3. Create Texture Object (with descriptors)
	hipCreateTextureObject(&m_texObj, &resDesc, &texDesc, NULL);
}

const int CDelaunay_2D_Cuda_XGraph_Adaptor::Dim = 2;
const int CDelaunay_2D_Cuda_XGraph_Adaptor::NCellVertices = 3;
const int CDelaunay_2D_Cuda_XGraph_Adaptor::NCellNeighbors = 3;