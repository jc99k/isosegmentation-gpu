#include "hip/hip_runtime.h"
#include "Image3D.h"

CImage_3D_Cuda_XGraph_Adaptor::~CImage_3D_Cuda_XGraph_Adaptor()
{
	//hipFree(m_updatepatterns);
	hipFree(m_sortedcells);
	hipFree(m_labels);
	hipFree(m_colorpatterns);
	hipFree(m_similarities);
}

void CImage_3D_Cuda_XGraph_Adaptor::UnifiedMalloc()
{
	//hipMallocManaged(&m_updatepatterns, (v)*sizeof(int));
	hipMallocManaged(&m_sortedcells, (v)*sizeof(int));
	hipMallocManaged(&m_labels, (v)*sizeof(int));
	hipMallocManaged(&m_colorpatterns, (v)*sizeof(float));
	//hipMallocManaged(&m_similarities, (e)*sizeof(float));

	hipMallocManaged(&m_similarities, (v)*sizeof(SimilarityType));
}

void CImage_3D_Cuda_XGraph_Adaptor::Synchronize()
{
	hipDeviceSynchronize();
}

void CImage_3D_Cuda_XGraph_Adaptor::SetTexture(ImgType* imgdata, int width, int height, int depth)
{
	m_image_dimensions = make_int3(width, height, depth);
	hipMemcpy(m_colorpatterns, imgdata, m_vol*sizeof(ImgType), hipMemcpyHostToHost);
	m_pointcoords = m_colorpatterns;
}

void CImage_3D_Cuda_XGraph_Adaptor::FreeTexture()
{
	//hipDestroyTextureObject(m_texObj);
}

const int CImage_3D_Cuda_XGraph_Adaptor::Dim = 3;
const int CImage_3D_Cuda_XGraph_Adaptor::NCellVertices = 1;
const int CImage_3D_Cuda_XGraph_Adaptor::NCellNeighbors = 26;
const int CImage_3D_Cuda_XGraph_Adaptor::NFacetVertices = 2;
const int CImage_3D_Cuda_XGraph_Adaptor::NGraphColors = 8;