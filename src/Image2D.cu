#include "hip/hip_runtime.h"
#include "Image2D.h"

CImage_2D_Cuda_XGraph_Adaptor::~CImage_2D_Cuda_XGraph_Adaptor()
{
	//hipFree(m_updatepatterns);
	hipFree(m_sortedcells);
	hipFree(m_labels);
	hipFree(m_colorpatterns);
	hipFree(m_similarities);
}

void CImage_2D_Cuda_XGraph_Adaptor::UnifiedMalloc()
{
	//hipMallocManaged(&m_updatepatterns, (v)*sizeof(int));
	hipMallocManaged(&m_sortedcells, (v)*sizeof(int));
	hipMallocManaged(&m_labels, (v)*sizeof(int));
	hipMallocManaged(&m_colorpatterns, (v)*sizeof(float));
	//hipMallocManaged(&m_similarities, (e)*sizeof(float));

	hipMallocManaged(&m_similarities, (v)*sizeof(SimilarityType));
}

void CImage_2D_Cuda_XGraph_Adaptor::Synchronize()
{
	hipDeviceSynchronize();
}

void CImage_2D_Cuda_XGraph_Adaptor::SetTexture(ImgType* imgdata, int width, int height)
{
	m_image_dimensions = make_int3(width, height, 1);
	hipMemcpy(m_colorpatterns, imgdata, m_vol*sizeof(ImgType), hipMemcpyHostToHost);
	m_pointcoords = m_colorpatterns;
}

void CImage_2D_Cuda_XGraph_Adaptor::FreeTexture()
{
	//hipDestroyTextureObject(m_texObj);
}

const int CImage_2D_Cuda_XGraph_Adaptor::Dim = 2;
const int CImage_2D_Cuda_XGraph_Adaptor::NCellVertices = 1;
const int CImage_2D_Cuda_XGraph_Adaptor::NCellNeighbors = 8;
const int CImage_2D_Cuda_XGraph_Adaptor::NFacetVertices = 2;
const int CImage_2D_Cuda_XGraph_Adaptor::NGraphColors = 4;