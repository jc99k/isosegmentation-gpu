#include "hip/hip_runtime.h"
#include "Delaunay3D.cuh"

CDelaunay_3D_Cuda_XGraph_Adaptor::~CDelaunay_3D_Cuda_XGraph_Adaptor()
{
	hipFree(m_xadj);
	hipFree(m_adjncy);
	hipFree(m_updatepatterns);
	hipFree(m_labels);
	hipFree(m_colorpatterns);
	hipFree(m_similarities);
	hipFree(m_cellvertices);
	hipFree(m_delaunayvertices);

	hipDestroyTextureObject(m_texObj);
	if(d_volumeArray) hipFreeArray(d_volumeArray);
}

void CDelaunay_3D_Cuda_XGraph_Adaptor::UnifiedMalloc()
{
	hipMallocManaged(&m_xadj, (v + 1)*sizeof(int));
	hipMallocManaged(&m_adjncy, (e * 2)*sizeof(int));
	hipMallocManaged(&m_updatepatterns, (v)*sizeof(int));
	hipMallocManaged(&m_labels, (v)*sizeof(int));
	hipMallocManaged(&m_colorpatterns, (v)*sizeof(float));
	hipMallocManaged(&m_similarities, (e * 2)*sizeof(float));
	hipMallocManaged(&m_cellvertices, (v)*sizeof(int4));
	hipMallocManaged(&m_delaunayvertices, (dv)*sizeof(float3));
}

void CDelaunay_3D_Cuda_XGraph_Adaptor::Synchronize()
{
	hipDeviceSynchronize();
}

void CDelaunay_3D_Cuda_XGraph_Adaptor::SetTexture(ImgType* imgdata, int width, int height, int depth)
{
	m_image_dimensions = make_int3(width, height, depth);

	// CUDA MALLOC 3D ARRAY

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<ImgType>();
	hipExtent volumeSize = make_hipExtent(width, height, depth);
	hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize);

	// CUDA MEMCPY3D : Pitched-imgdata (Host) -> d_volumeArray (Device CUDA Array)

	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(imgdata, width*sizeof(ImgType), height, depth);
	copyParams.dstArray = d_volumeArray;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// Texture (new way) : Create texture object
	// 1. Resource Descriptor (resource to texture with)
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = d_volumeArray;

	// 2. Texture Descriptor (texturing settings)
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.normalizedCoords = false;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.addressMode[2] = hipAddressModeClamp;
	texDesc.readMode = hipReadModeElementType;

	// 3. Create Texture Object (with descriptors)
	hipCreateTextureObject(&m_texObj, &resDesc, &texDesc, NULL);

	printf("FINISHED TEXTURE ALLOC\n");
}

const int CDelaunay_3D_Cuda_XGraph_Adaptor::Dim = 3;
const int CDelaunay_3D_Cuda_XGraph_Adaptor::NCellVertices = 4;
const int CDelaunay_3D_Cuda_XGraph_Adaptor::NCellNeighbors = 4;