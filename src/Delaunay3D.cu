#include "hip/hip_runtime.h"
#include "Delaunay3D.h"

CDelaunay_3D_Cuda_XGraph_Adaptor::~CDelaunay_3D_Cuda_XGraph_Adaptor()
{
	hipFree(m_pointcoords);
	hipFree(m_cellvertices);
	//hipFree(m_nodeedges);
	//hipFree(m_updatepatterns);
	hipFree(m_sortedcells);
	hipFree(m_colorpatterns);
	hipFree(m_labels);
	hipFree(m_sizes);
	//hipFree(m_facetvertices);
	//hipFree(m_edgenodes);
	//hipFree(m_similarities);
	
	hipFree(m_neighbors);
	hipFree(m_similarities);
	FreeTexture();
}

void CDelaunay_3D_Cuda_XGraph_Adaptor::UnifiedMalloc()
{
	hipMallocManaged(&m_pointcoords, (dv)*sizeof(PointCoordsType));
	hipMallocManaged(&m_cellvertices, (v)*sizeof(CellVerticesType));
	//hipMallocManaged(&m_nodeedges, (v)*sizeof(NodeEdgesType));
	//hipMallocManaged(&m_updatepatterns, (v)*sizeof(int));
	hipMallocManaged(&m_sortedcells, (v)*sizeof(int));
	hipMallocManaged(&m_colorpatterns, (v)*sizeof(float));
	hipMallocManaged(&m_labels, (v)*sizeof(int));
	hipMallocManaged(&m_sizes, (v)*sizeof(float));
	//hipMallocManaged(&m_facetvertices, (e)*sizeof(FacetsType));
	//hipMallocManaged(&m_edgenodes, (e)*sizeof(EdgeNodesType));
	//hipMallocManaged(&m_similarities, (e)*sizeof(float));

	hipMallocManaged(&m_neighbors, (v)*sizeof(NeighborsType));
	hipMallocManaged(&m_similarities, (v)*sizeof(SimilarityType));
}

void CDelaunay_3D_Cuda_XGraph_Adaptor::Synchronize()
{
	hipDeviceSynchronize();
}

void CDelaunay_3D_Cuda_XGraph_Adaptor::SetTexture(ImgType* imgdata, int width, int height, int depth)
{
	m_image_dimensions = make_int3(width, height, depth);
	m_vol = width*height*depth;
	// CUDA MALLOC 3D ARRAY

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<ImgType>();
	hipExtent volumeSize = make_hipExtent(width, height, depth);
	hipMalloc3DArray(&m_texdata, &channelDesc, volumeSize);

	// CUDA MEMCPY3D : Pitched-imgdata (Host) -> m_texdata (Device CUDA Array)

	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(imgdata, width*sizeof(ImgType), width, height);
	copyParams.dstArray = m_texdata;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	
	hipMemcpy3D(&copyParams);

	// Texture (new way) : Create texture object
	// 1. Resource Descriptor (resource to texture with)
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = m_texdata;

	// 2. Texture Descriptor (texturing settings)
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.normalizedCoords = false;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = hipAddressModeBorder;
	texDesc.addressMode[1] = hipAddressModeBorder;
	texDesc.addressMode[2] = hipAddressModeBorder;
	texDesc.readMode = hipReadModeElementType;

	// 3. Create Texture Object (with descriptors)
	hipCreateTextureObject(&m_texObj, &resDesc, &texDesc, NULL);

	//printf("Texture memory ready!\n");
}

void CDelaunay_3D_Cuda_XGraph_Adaptor::FreeTexture()
{
	hipDestroyTextureObject(m_texObj);
	hipFreeArray(m_texdata);
}

const int CDelaunay_3D_Cuda_XGraph_Adaptor::Dim = 3;
const int CDelaunay_3D_Cuda_XGraph_Adaptor::NCellVertices = 4;
const int CDelaunay_3D_Cuda_XGraph_Adaptor::NCellNeighbors = 4;
const int CDelaunay_3D_Cuda_XGraph_Adaptor::NFacetVertices = 3;
const int CDelaunay_3D_Cuda_XGraph_Adaptor::NGraphColors = 5;