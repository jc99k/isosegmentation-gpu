#include "hip/hip_runtime.h"
#include "img_reader.h"

using namespace std;
using namespace cimg_library;

template<class G>
CImageReader<G>::CImageReader(float* imgdata, hipTextureObject_t& texObj) : m_imagedata(imgdata), m_ptexObject(&texObj) {}

template<class G>
void CImageReader<G>::load_file()
{
	string filename;
	cout << "Enter Image filename: ";
	cin >> filename;
	ifstream is(filename, ifstream::in);
	CImg<float> image(filename.c_str());
	//image = image.RGBtoLab().channel(0).normalize(0, 255);

	m_imageextent = make_int3(image.width(), image.height(), image.depth());
	size_t pitch, widthInBytes = image.width()*sizeof(float);

	hipMallocPitch(&m_imagedata, &pitch, widthInBytes, image.height());
	hipMemcpy2D(m_imagedata, pitch, image.data(), widthInBytes, widthInBytes, image.height(), hipMemcpyHostToDevice);

	// Texture (new way) : Create texture object
	// 1. Resource Descriptor (resource to texture with)
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = m_imagedata;
	resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
	resDesc.res.pitch2D.width = image.width();
	resDesc.res.pitch2D.height = image.height();
	resDesc.res.pitch2D.pitchInBytes = pitch;

	// 2. Texture Descriptor (texturing settings)
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
	texDesc.filterMode = hipFilterModeLinear;      // linear interpolation
	texDesc.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.addressMode[2] = hipAddressModeWrap; // 3D
	
	// 3. Create Texture Object (with descriptors)
	hipCreateTextureObject(m_ptexObject, &resDesc, &texDesc, NULL);
}

template<class G>
int3& CImageReader<G>::image_extent() { return m_imageextent; }


template class CImageReader<int>;